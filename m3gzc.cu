#include"m3gzc_kernel.cu"
#include"m3gzc.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>

using namespace std;

bool InitGPUSet()
{
    char GPU[100] = "GPU: ";
    hipDeviceProp_t tCard;
    int num = 0;
    if(hipSuccess == hipGetDeviceCount(&num))
    {
        for(int i = 0; i < num; ++ i)
        {
            hipSetDevice(i);
            hipGetDeviceProperties(&tCard, i);
            puts(tCard.name);
        }
    }
    else  return false;
    return true;
}

bool cuPrintInit()
{
    hipError_t err = cudaPrintfInit();
    if(0 != strcmp("no error", hipGetErrorString(err)))  return false;
    return true;
}

void reportError(){
    hipError_t cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess) 
        printf("error \"%s\".\n", hipGetErrorString(cudaerr));
    else printf("success\n");
}

int m3gzc(SerializedSampleSet sss1,SerializedSampleSet sss2){

    sss1.print();
    sss2.print();

    if(!InitGPUSet()) {
        puts("Device is not ready\n");
        exit(0);
    }
    hipSetDevice(0);
    if(!cuPrintInit()) {
        puts("cuPrint init failed\n");
        exit(0);
    }
    
    cudaPrintfDisplay(stdout,true);

    Data_Node * test_data=new Data_Node,*d_test_data;
    test_data->index=1;
    test_data->value=1;

    int test_data_length=1;
    int * d_test_data_length;

    int resultSize=sss1.numSample*sss2.numSample;//SerializedSampleSet::max_num_sample*SerializedSampleSet::max_num_sample;
    float *resultMat=new float[resultSize];
    float * d_resultMat;
    int BLOCK_SIZE=8;

    SerializedSampleSet *d_sss1,*d_sss2;

    size_t sssSize=sizeof(SerializedSampleSet);

    hipMalloc(&d_sss1,sssSize);
    hipMalloc(&d_sss2,sssSize);
    hipMalloc(&d_resultMat,resultSize*sizeof(float));
    hipMalloc((void**)&d_test_data,test_data_length*sizeof(Data_Node));
    hipMalloc(&d_test_data_length,sizeof(int));

    hipMemcpy(d_sss1,&sss1,sssSize,hipMemcpyHostToDevice);
    hipMemcpy(d_sss2,&sss2,sssSize,hipMemcpyHostToDevice);
    hipMemcpy((void*)d_test_data,test_data,test_data_length*sizeof(Data_Node),hipMemcpyHostToDevice);
    hipMemcpy(d_test_data_length,&test_data_length,sizeof(int),hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
    dim3 dimGrid((sss1.numSample-1)/BLOCK_SIZE+1,(sss2.numSample-1)/BLOCK_SIZE+1);

    {
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) 
            printf("before kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
        else printf("before kernel launch success\n");
    }

    m3gzcKernel<<<dimGrid,dimBlock>>>(d_test_data,d_test_data_length,d_sss1,d_sss2,d_resultMat);

    //reportError();

    {
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) 
            printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
        else printf("kernel launch success\n");
    }

    hipMemcpy(resultMat,d_resultMat,resultSize*sizeof(float),hipMemcpyDeviceToHost);

    for(int i=0;i<resultSize;i++){
        if((i)%sss2.numSample==0) cout<<endl;
        cout<<resultMat[i]<<'\t';
    }
    cout<<endl;

    cudaPrintfEnd();
}


