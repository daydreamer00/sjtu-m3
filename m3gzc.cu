#include"m3gzc_kernel.cu"
#include"m3gzc.h"
#include<hip/hip_runtime.h>
#include<stdio.h>

int m3gzc(SerializedSampleSet sss1,SerializedSampleSet sss2){

    cudaPrintfInit();
    cudaPrintfDisplay(stdout,true);

    Data_Node * test_data=new Data_Node,*d_test_data;
    test_data->index=1;
    test_data->value=3;

    int test_data_length=1;
    int * d_test_data_length;

    int resultSize=SerializedSampleSet::max_num_sample*SerializedSampleSet::max_num_sample;
    int *resultMat=new int[resultSize];
    int * d_resultMat;
    int BLOCK_SIZE=16;

    SerializedSampleSet *d_sss1,*d_sss2;

    size_t sssSize=sizeof(SerializedSampleSet);

    hipMalloc(&d_sss1,sssSize);
    hipMalloc(&d_sss2,sssSize);
    hipMalloc(&d_resultMat,resultSize*sizeof(int));
    hipMalloc((void**)&d_test_data,test_data_length*sizeof(Data_Node));
    hipMalloc(&d_test_data_length,sizeof(int));

    hipMemcpy(d_sss1,&sss1,sssSize,hipMemcpyHostToDevice);
    hipMemcpy(d_sss2,&sss2,sssSize,hipMemcpyHostToDevice);
    hipMemcpy((void*)d_test_data,test_data,test_data_length*sizeof(Data_Node),hipMemcpyHostToDevice);
    hipMemcpy(d_test_data_length,&test_data_length,sizeof(int),hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
    dim3 dimGrid(sss1.numSample/BLOCK_SIZE,sss2.numSample/BLOCK_SIZE);
    m3gzcKernel<<<dimGrid,dimBlock>>>(d_test_data,d_test_data_length,d_sss1,d_sss2,d_resultMat);

    hipMemcpy(resultMat,d_resultMat,resultSize*sizeof(int),hipMemcpyDeviceToHost);

    cudaPrintfEnd();
}


