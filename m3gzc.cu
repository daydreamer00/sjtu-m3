#include "hip/hip_runtime.h"
#include"m3gzc_kernel.cu"
#include"m3gzc.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>

using namespace std;

bool InitGPUSet()
{
    char GPU[100] = "GPU: ";
    hipDeviceProp_t tCard;
    int num = 0;
    if(hipSuccess == hipGetDeviceCount(&num))
    {
        for(int i = 0; i < num; ++ i)
        {
            hipSetDevice(i);
            hipGetDeviceProperties(&tCard, i);
            puts(tCard.name);
        }
    }
    else  return false;
    return true;
}

bool cuPrintInit()
{
    hipError_t err = cudaPrintfInit();
    if(0 != strcmp("no error", hipGetErrorString(err)))  return false;
    return true;
}

void reportError(){
    hipError_t cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess) 
        printf("error \"%s\".\n", hipGetErrorString(cudaerr));
    //else printf("success\n");
}

int *m3gzcGPU(SerializedSampleSet sss1,SerializedSampleSet sss2){
    clock_t timer;
    clock_t timer0;
    float time;
    TIMER_BEGIN(timer);
    TIMER_BEGIN(timer0);
    hipEvent_t ev0,ev1;

    hipEventCreate(&ev0);
    hipEventCreate(&ev1);
    
    //sss1.print();
    //sss2.print();

    hipSetDevice(0);

    hipEventRecord(ev0,0);

    hipError_t cudaerr;
    int attr=0;

    cudaerr=hipDeviceGetAttribute(&attr,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
    if (cudaerr != hipSuccess) 
        printf("error \"%s\".\n", hipGetErrorString(cudaerr));
    //cout<<"Max shared mem per block(bytes): "<<attr<<endl;

    Data_Node * test_data=new Data_Node,*d_test_data;
    test_data->index=1;
    test_data->value=1;

    int test_data_length=1;
    int * d_test_data_length;

    int resultSize=sss1.numSample*sss2.numSample;//SerializedSampleSet::max_num_sample*SerializedSampleSet::max_num_sample;
    float *resultMat=new float[resultSize];
    float * d_resultMat;
    //int BLOCK_SIZE=16;

    SerializedSampleSet *d_sss1,*d_sss2;

    size_t sssSize=sizeof(SerializedSampleSet);
    TIMER_PRINT("pre",timer);


    TIMER_BEGIN(timer);
    hipMalloc(&d_sss1,sssSize);
    hipMalloc(&d_sss2,sssSize);
    hipMalloc(&d_resultMat,resultSize*sizeof(float));
    hipMalloc((void**)&d_test_data,test_data_length*sizeof(Data_Node));
    hipMalloc(&d_test_data_length,sizeof(int));
    TIMER_PRINT("malloc",timer);

    TIMER_BEGIN(timer);
    hipMemcpy(d_sss1,&sss1,sssSize,hipMemcpyHostToDevice);
    hipMemcpy(d_sss2,&sss2,sssSize,hipMemcpyHostToDevice);
    hipMemcpy((void*)d_test_data,test_data,test_data_length*sizeof(Data_Node),hipMemcpyHostToDevice);
    hipMemcpy(d_test_data_length,&test_data_length,sizeof(int),hipMemcpyHostToDevice);
    TIMER_PRINT("mcopy",timer);

    TIMER_BEGIN(timer);
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
    dim3 dimGrid((sss1.numSample-1)/BLOCK_SIZE+1,(sss2.numSample-1)/BLOCK_SIZE+1);

    TIMER_PRINT("pre gzc compute",timer);

    TIMER_BEGIN(timer);
    {
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) 
            printf("before kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
        //else printf("before kernel launch success\n");
    }
    TIMER_PRINT("error check",timer);

    TIMER_BEGIN(timer);
    m3gzcKernel<<<dimGrid,dimBlock>>>(d_test_data,d_test_data_length,d_sss1,d_sss2,d_resultMat);
    //m3gzcKernelWithSharedMemory<<<dimGrid,dimBlock>>>(d_test_data,d_test_data_length,d_sss1,d_sss2,d_resultMat);
    TIMER_PRINT("gzc compute",timer);
    TIMER_BEGIN(timer);
    reportError();

    {
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) 
            printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
        //else printf("kernel launch success\n");
    }
    TIMER_PRINT("error check",timer);

    TIMER_BEGIN(timer);
    hipMemcpy(resultMat,d_resultMat,resultSize*sizeof(float),hipMemcpyDeviceToHost);

    TIMER_PRINT("mcopy",timer);
    //for(int i=0;i<resultSize;i++){
    //    if((i)%sss2.numSample==0) cout<<endl;
    //    cout<<resultMat[i]<<'\t';
    //}
    //cout<<endl;

    //for(int i=0;i<resultSize;i++){
    //    if((i)%sss2.numSample==0 && i>0) cout<<endl;
    //    cout<<resultMat[i]<<'\t';
    //}

    TIMER_BEGIN(timer);
    int *resultArray= new int[sss1.numSample];
    int *d_resultArray;
    hipMalloc(&d_resultArray,sss1.numSample*sizeof(int));

    int threadsPerBlock=128;
    int blockPerGrid=(sss1.numSample-1)/threadsPerBlock+1;
    TIMER_PRINT("malloc",timer);

    TIMER_BEGIN(timer);
    {
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) 
            printf("before kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
        //else printf("before kernel launch success\n");
    }
    TIMER_PRINT("error check",timer);
    TIMER_BEGIN(timer);
    minmaxKernel<<<blockPerGrid,threadsPerBlock>>>(d_resultMat,sss1.numSample,sss2.numSample,d_resultArray);
    TIMER_PRINT("min kernel",timer);

    TIMER_BEGIN(timer);
    reportError();

    {
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) 
            printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
        //else printf("kernel launch success\n");
    }
    TIMER_PRINT("error check",timer);

    TIMER_BEGIN(timer);
    hipMemcpy(resultArray,d_resultArray,sss1.numSample*sizeof(int),hipMemcpyDeviceToHost);
    TIMER_PRINT("post copy",timer);

    //cout<<sss1.numSample<<' '<<sss2.numSample<<endl;
    //for(int i=0;i<sss1.numSample;i++){
    //    cout<<i<<' '<<resultArray[i]<<endl;
    //}
    //cout<<endl;

    TIMER_BEGIN(timer);

    hipFree(d_sss1);
    hipFree(d_sss2);
    hipFree(d_resultMat);
    hipFree(d_test_data);
    hipFree(d_test_data_length);
    hipFree(d_resultArray);

    delete test_data;

    hipEventRecord(ev1,0);
    hipEventSynchronize(ev0);
    hipEventSynchronize(ev1);

    hipEventElapsedTime(&time,ev0,ev1);
    cout<<"cuda event time "<<time<<endl;

    hipEventDestroy(ev0);
    hipEventDestroy(ev1);

    TIMER_PRINT("post compute",timer);
    TIMER_PRINT("total compute time",timer0);
    return resultArray;
}

float getDistance2(Data_Sample test_sample,const SerializedSampleSet sss,int i){
    int dataIndexBegin,dataIndexEnd;
    dataIndexBegin=i==0?0:sss.dataNodeOffsetArray[i-1];
    dataIndexEnd=sss.dataNodeOffsetArray[i];
    int j=0,k=dataIndexBegin;
    float sum=0;
    //cout<<endl<<dataIndexBegin<<endl<<dataIndexEnd<<endl;
    while(1){
        //cout<<j<<endl<<k<<endl;
        if(j==test_sample.data_vector_length && k==dataIndexEnd) break;
        if(k==dataIndexEnd) {
            sum+=SQUARE(test_sample.data_vector[j].value);
            j++;
        } else if(j==test_sample.data_vector_length) {
            sum+=SQUARE(sss.dataNodeValueArray[k]);
            k++;
        } else if(test_sample.data_vector[j].index < sss.dataNodeIndexArray[k]){
            sum+=SQUARE(test_sample.data_vector[j].value);
            j++;
        } else if(test_sample.data_vector[j].index > sss.dataNodeIndexArray[k]){
            sum+=SQUARE(sss.dataNodeValueArray[k]);
            k++;
        } else {
            float tmp=test_sample.data_vector[j].value-sss.dataNodeValueArray[k];
            sum+=SQUARE(tmp);
            //cout<<sum<<endl;
            j++;
            k++;
        }
    }
    return sum;
}

float getDistance2(const SerializedSampleSet sss1,int i,const SerializedSampleSet sss2,int j){
    int dataIndexBegin1,dataIndexEnd1;
    dataIndexBegin1=i==0?0:sss1.dataNodeOffsetArray[i-1];
    dataIndexEnd1=sss1.dataNodeOffsetArray[i];
    int dataIndexBegin2,dataIndexEnd2;
    dataIndexBegin2=j==0?0:sss2.dataNodeOffsetArray[j-1];
    dataIndexEnd2=sss2.dataNodeOffsetArray[j];
    int ii=dataIndexBegin1,ij=dataIndexBegin2;
    float sum=0;
    while (1){
        if(ii==dataIndexEnd1 && ij==dataIndexEnd2) break;
        else if(ii==dataIndexEnd1) {
            sum+=SQUARE(sss2.dataNodeValueArray[ij]);
            ij++;
        }else if (ij==dataIndexEnd2) {
            sum+=SQUARE(sss1.dataNodeValueArray[ii]);
            ii++;
        }else if (sss1.dataNodeIndexArray[ii]<sss2.dataNodeIndexArray[ij]){
            sum+=SQUARE(sss1.dataNodeValueArray[ii]);
            ii++;
        }else if (sss1.dataNodeIndexArray[ii]>sss2.dataNodeIndexArray[ij]){
            sum+=SQUARE(sss2.dataNodeValueArray[ij]);
            ij++;
        } else {
            float tmp=sss2.dataNodeValueArray[ij]-sss1.dataNodeValueArray[ii];
            sum+=SQUARE(tmp);
            ij++;
            ii++;
        } 
    }
    return sum;
}


int *m3gzcCPU(SerializedSampleSet sss1,SerializedSampleSet sss2){
    //sss1.print();
    //sss2.print();

    float * sumArray1,*sumArray2;
    int *resultArray;
    sumArray1=new float[sss1.numSample];
    sumArray2=new float[sss2.numSample];
    resultArray=new int[sss1.numSample];
    
    Data_Node * test_data=new Data_Node;
    test_data->index=1;
    test_data->value=1;

    Data_Sample test_sample;
    test_sample.index=0;
    test_sample.label=0;
    test_sample.data_vector_length=1;
    test_sample.data_vector=test_data;

    for(int i=0;i<sss1.numSample;i++) sumArray1[i]=getDistance2(test_sample,sss1,i);
    cout<<sumArray1[0]<<endl;
    for(int i=0;i<sss2.numSample;i++) sumArray2[i]=getDistance2(test_sample,sss2,i);

    for(int i=0;i<sss1.numSample;i++){
        float min=1;
        for(int j=0;j<sss2.numSample;j++) {
            float sum0=getDistance2(sss1,i,sss2,j);
            //cout<<i<<' '<<j<<' '<<sum0<<endl;
            float v=exp(-4*sumArray1[i]/sum0)-exp(-4*sumArray2[j]/sum0);
            //cout<<i<<' '<<j<<' '<<v<<endl;
            if (v<min) min=v;
        }

        //cout<<i<<' '<<min<<endl;

        if(min>THRESHOLD) resultArray[i]=1;
        else if(min<-THRESHOLD) resultArray[i]=-1;
        else resultArray[i]=0;
    }

    delete test_sample.data_vector;
    return resultArray;
}
