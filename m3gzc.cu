#include "hip/hip_runtime.h"
#include"m3gzc_kernel.cu"
#include"m3gzc.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>

using namespace std;

bool InitGPUSet()
{
    char GPU[100] = "GPU: ";
    hipDeviceProp_t tCard;
    int num = 0;
    if(hipSuccess == hipGetDeviceCount(&num))
    {
        for(int i = 0; i < num; ++ i)
        {
            hipSetDevice(i);
            hipGetDeviceProperties(&tCard, i);
            puts(tCard.name);
        }
    }
    else  return false;
    return true;
}

bool cuPrintInit()
{
    hipError_t err = cudaPrintfInit();
    if(0 != strcmp("no error", hipGetErrorString(err)))  return false;
    return true;
}

void reportError(){
    hipError_t cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess) 
        printf("error \"%s\".\n", hipGetErrorString(cudaerr));
    else printf("success\n");
}

int *m3gzcGPU(SerializedSampleSet sss1,SerializedSampleSet sss2){

    sss1.print();
    sss2.print();

    hipSetDevice(0);

    hipError_t cudaerr;
    int attr=0;

    cudaerr=hipDeviceGetAttribute(&attr,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
    if (cudaerr != hipSuccess) 
        printf("error \"%s\".\n", hipGetErrorString(cudaerr));
    cout<<"Max shared mem per block(bytes): "<<attr<<endl;

    Data_Node * test_data=new Data_Node,*d_test_data;
    test_data->index=1;
    test_data->value=1;

    int test_data_length=1;
    int * d_test_data_length;

    int resultSize=sss1.numSample*sss2.numSample;//SerializedSampleSet::max_num_sample*SerializedSampleSet::max_num_sample;
    float *resultMat=new float[resultSize];
    float * d_resultMat;
    //int BLOCK_SIZE=16;

    SerializedSampleSet *d_sss1,*d_sss2;

    size_t sssSize=sizeof(SerializedSampleSet);

    hipMalloc(&d_sss1,sssSize);
    hipMalloc(&d_sss2,sssSize);
    hipMalloc(&d_resultMat,resultSize*sizeof(float));
    hipMalloc((void**)&d_test_data,test_data_length*sizeof(Data_Node));
    hipMalloc(&d_test_data_length,sizeof(int));

    hipMemcpy(d_sss1,&sss1,sssSize,hipMemcpyHostToDevice);
    hipMemcpy(d_sss2,&sss2,sssSize,hipMemcpyHostToDevice);
    hipMemcpy((void*)d_test_data,test_data,test_data_length*sizeof(Data_Node),hipMemcpyHostToDevice);
    hipMemcpy(d_test_data_length,&test_data_length,sizeof(int),hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
    dim3 dimGrid((sss1.numSample-1)/BLOCK_SIZE+1,(sss2.numSample-1)/BLOCK_SIZE+1);

    {
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) 
            printf("before kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
        else printf("before kernel launch success\n");
    }

    //m3gzcKernel<<<dimGrid,dimBlock>>>(d_test_data,d_test_data_length,d_sss1,d_sss2,d_resultMat);
    m3gzcKernelWithSharedMemory<<<dimGrid,dimBlock>>>(d_test_data,d_test_data_length,d_sss1,d_sss2,d_resultMat);

    reportError();

    {
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) 
            printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
        else printf("kernel launch success\n");
    }

    hipMemcpy(resultMat,d_resultMat,resultSize*sizeof(float),hipMemcpyDeviceToHost);

    //for(int i=0;i<resultSize;i++){
    //    if((i)%sss2.numSample==0) cout<<endl;
    //    cout<<resultMat[i]<<'\t';
    //}
    //cout<<endl;

    //for(int i=0;i<resultSize;i++){
    //    if((i)%sss2.numSample==0 && i>0) cout<<endl;
    //    cout<<resultMat[i]<<'\t';
    //}

    int *resultArray= new int[sss1.numSample];
    int *d_resultArray;
    hipMalloc(&d_resultArray,sss1.numSample*sizeof(int));

    int threadsPerBlock=128;
    int blockPerGrid=(sss1.numSample-1)/threadsPerBlock+1;

    {
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) 
            printf("before kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
        else printf("before kernel launch success\n");
    }
    minmaxKernel<<<blockPerGrid,threadsPerBlock>>>(d_resultMat,sss1.numSample,sss2.numSample,d_resultArray);

    reportError();

    {
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) 
            printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
        else printf("kernel launch success\n");
    }

    hipMemcpy(resultArray,d_resultArray,sss1.numSample*sizeof(int),hipMemcpyDeviceToHost);

    for(int i=0;i<sss1.numSample;i++){
        cout<<resultArray[i]<<' ';
    }
    cout<<endl;

    hipFree(d_sss1);
    hipFree(d_sss2);
    hipFree(d_resultMat);
    hipFree(d_test_data);
    hipFree(d_test_data_length);
    hipFree(d_resultArray);

    delete test_data;

    return resultArray;
}

float getDistance2(Data_Sample test_sample,const SerializedSampleSet sss,int i){
    int dataIndexBegin,dataIndexEnd;
    dataIndexBegin=i==0?0:sss.dataNodeIndexArray[i-1];
    dataIndexEnd=dataNodeIndexArray[i];
    int j=0,k=dataIndexBegin;
    float sum=0;
    while(1){
        if(j==test_sample.data_vector_length && k==dataIndexEnd) break;
        if(k==dataIndexEnd) {
            sum+=SQUARE(test_sample.data_vector[j].value);
            j++;
        } else if(j==test_sample.data_vector_length) {
            sum+=SQUARE(sss.dataNodeValueArray[k]);
            k++;
        } if(test_sample.data_vector[j].index < sss.dataNodeIndexArray[k]){
            sum+=SQUARE(test_sample.data_vector[j].value);
            j++;
        } else if(test_sample.data_vector[j].index < sss.dataNodeIndexArray[k]){
            sum+=SQUARE(sss.dataNodeValueArray[k]);
            k++;
        } else {
            sum+=SQUARE(test_sample.data_vector[j].value)+SQUARE(sss.dataNodeValueArray[k]);
            j++;
            k++;
        }
    }
    return sum;
}


int *m3gzcCPU(SerializedSampleSet sss1,SerializedSampleSet sss2){
    float * sumArray1,*sumArray2,*resultArray;
    sumArray1=new float[sss1.numSample];
    sumArray2=new float[sss2.numSample];
    resultArray=new float[sss1.numSample];
    
    Data_Node * test_data=new Data_Node,*d_test_data;
    test_data->index=1;
    test_data->value=1;

    Data_Sample test_sample;
    test_sample.index=0;
    test_sample.label=0;
    test_sample.data_vector_length=1;
    test_sample.data_vector=test_data;

    for(int i=0;i<sss1.numSample;i++) sumArray1[i]=getDistance2(test_sample,sss1,i);
    for(int i=0;i<sss2.numSample;i++) sumArray2[i]=getDistance2(test_sample,sss2,i);

    for(int i=0;i<sss1.numSample;i++){
        float min=1;
        for(int j=0;j<sss2.numSample;j++) {
            float v=
            
    }

    delete test_sample.data_vector;
    delete test_sample;
}

