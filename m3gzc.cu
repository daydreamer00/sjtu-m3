#include "hip/hip_runtime.h"
#include"m3gzc_kernel.cu"
#include"m3gzc.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>
#include<fstream>
#include<string>

using namespace std;

void reportError(){
    hipError_t cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess) {
        printf("error \"%s\".\n", hipGetErrorString(cudaerr));
        exit(1);
    }
    //else printf("success\n");
}

void recordTime(hipEvent_t &evtBegin,hipEvent_t &evtEnd,string message){
    hipEventRecord(evtEnd,0);
    float time;
    hipEventSynchronize(evtBegin);
    hipEventSynchronize(evtEnd);
    hipEventElapsedTime(&time,evtBegin,evtEnd);
    fbreakdown<<message<<'\t'<<time<<endl;
    hipEventRecord(evtBegin,0);
}

int *m3gzcGPU(SerializedSampleSet sss1,SerializedSampleSet sss2,SerializedSampleSet sss3){
    clock_t timer;
    clock_t timer0;
    float time;
    TIMER_BEGIN(timer);
    TIMER_BEGIN(timer0);
    hipEvent_t ev0,ev1,ev2,ev3;

    hipEventCreate(&ev0);
    hipEventCreate(&ev1);
    hipEventCreate(&ev2);
    hipEventCreate(&ev3);
    
    //sss1.print();
    //sss2.print();
    //sss3.print();

    hipSetDevice(0);

    hipEventRecord(ev0,0);
    hipEventRecord(ev2,0);

    hipError_t cudaerr;
    int attr=0;

    cudaerr=hipDeviceGetAttribute(&attr,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
    if (cudaerr != hipSuccess) 
        printf("error \"%s\".\n", hipGetErrorString(cudaerr));
    //cout<<"Max shared mem per block(bytes): "<<attr<<endl;

    Data_Node * test_data=new Data_Node,*d_test_data;
    test_data->index=1;
    test_data->value=1;

    int test_data_length=1;
    int * d_test_data_length;

    int resultSize=sss1.numSample*sss2.numSample*sss3.numSample;//SerializedSampleSet::max_num_sample*SerializedSampleSet::max_num_sample;
    float *resultMat=new float[resultSize];
    float * d_resultMat;
    //int BLOCK_SIZE=16;

    SerializedSampleSet d_sss1,d_sss2,d_sss3;
    SerializedSampleSet *h_ssspArray[3],d_sssArray[3];
    h_ssspArray[0]=&sss1;
    h_ssspArray[1]=&sss2;
    h_ssspArray[2]=&sss3;

    size_t sssSize=sizeof(SerializedSampleSet);
    TIMER_PRINT("pre",timer);
    recordTime(ev2,ev3,"preprocessing 1");

    TIMER_BEGIN(timer);
    
    for(int i=0;i<3;i++){
        d_sssArray[i].numSample=h_ssspArray[i]->numSample;
        hipMalloc(&(d_sssArray[i].labelArray),sizeof(float)*MAX_NUM_SAMPLE);
        hipMalloc(&(d_sssArray[i].dataNodeOffsetArray),sizeof(int)*MAX_NUM_SAMPLE);
        hipMalloc(&(d_sssArray[i].dataNodeIndexArray),sizeof(int)*MAX_NUM_SAMPLE*AVERAGE_DATA_PER_SAMPLE);
        hipMalloc(&(d_sssArray[i].dataNodeValueArray),sizeof(float)*MAX_NUM_SAMPLE*AVERAGE_DATA_PER_SAMPLE);
        hipMemcpy(d_sssArray[i].labelArray,h_ssspArray[i]->labelArray,sizeof(float)*MAX_NUM_SAMPLE,hipMemcpyHostToDevice);
        hipMemcpy(d_sssArray[i].dataNodeOffsetArray,h_ssspArray[i]->dataNodeOffsetArray,sizeof(int)*MAX_NUM_SAMPLE,hipMemcpyHostToDevice);
        hipMemcpy(d_sssArray[i].dataNodeIndexArray,h_ssspArray[i]->dataNodeIndexArray,sizeof(int)*MAX_NUM_SAMPLE*AVERAGE_DATA_PER_SAMPLE,hipMemcpyHostToDevice);
        hipMemcpy(d_sssArray[i].dataNodeValueArray,h_ssspArray[i]->dataNodeValueArray,sizeof(float)*MAX_NUM_SAMPLE*AVERAGE_DATA_PER_SAMPLE,hipMemcpyHostToDevice);
    }

    //hipMalloc(&d_sss1,sssSize);
    //hipMalloc(&d_sss2,sssSize);
    //hipMalloc(&d_sss3,sssSize);
    hipMalloc(&d_resultMat,resultSize*sizeof(float));
    hipMalloc((void**)&d_test_data,test_data_length*sizeof(Data_Node));
    hipMalloc(&d_test_data_length,sizeof(int));

    recordTime(ev2,ev3,"Mem alloc and copy");
    TIMER_PRINT("malloc",timer);

    TIMER_BEGIN(timer);
    //hipMemcpy(d_sss1,&sss1,sssSize,hipMemcpyHostToDevice);
    //hipMemcpy(d_sss2,&sss2,sssSize,hipMemcpyHostToDevice);
    //hipMemcpy(d_sss3,&sss3,sssSize,hipMemcpyHostToDevice);
    //hipMemcpy((void*)d_test_data,test_data,test_data_length*sizeof(Data_Node),hipMemcpyHostToDevice);
    //hipMemcpy(d_test_data_length,&test_data_length,sizeof(int),hipMemcpyHostToDevice);
    TIMER_PRINT("mcopy",timer);

    TIMER_BEGIN(timer);
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
    dim3 dimGrid((sss1.numSample-1)/BLOCK_SIZE+1,(sss2.numSample-1)/BLOCK_SIZE+1);

    TIMER_PRINT("pre gzc compute",timer);

    TIMER_BEGIN(timer);
    {
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) 
            printf("before kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
        //else printf("before kernel launch success\n");
    }
    TIMER_PRINT("error check",timer);

    TIMER_BEGIN(timer);
    m3gzcKernel<<<dimGrid,dimBlock>>>(d_test_data,d_test_data_length,d_sss1,d_sss2,d_sss3,d_resultMat);
    //m3gzcKernelWithSharedMemory<<<dimGrid,dimBlock>>>(d_test_data,d_test_data_length,d_sssArray[0],d_sssArray[1],d_sssArray[2],d_resultMat);

    recordTime(ev2,ev3,"GZC computing(kernel 1)");
    TIMER_PRINT("gzc compute",timer);
    TIMER_BEGIN(timer);
    reportError();

    {
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) 
            printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
        //else printf("kernel launch success\n");
    }
    TIMER_PRINT("error check",timer);

    TIMER_BEGIN(timer);
    //hipMemcpy(resultMat,d_resultMat,resultSize*sizeof(float),hipMemcpyDeviceToHost);

    TIMER_PRINT("mcopy",timer);

    //cout<<resultMat[0]<<endl;
    //for(int i=0;i<resultSize;i++){
    //    if((i)%sss2.numSample==0 && i>0) cout<<endl;
    //    cout<<resultMat[i]<<'\t';
    //}

    TIMER_BEGIN(timer);
    resultSize=sss1.numSample*sss3.numSample;
    int *resultArray= new int[resultSize];
    int *d_resultArray;
    hipMalloc(&d_resultArray,resultSize*sizeof(int));

    int threadsPerBlock=128;
    int blockPerGrid=(sss1.numSample-1)/threadsPerBlock+1;

    dimBlock=dim3(BLOCK_SIZE,BLOCK_SIZE);
    dimGrid=dim3((sss1.numSample-1)/BLOCK_SIZE+1,(sss3.numSample-1)/BLOCK_SIZE+1);
    TIMER_PRINT("malloc",timer);

    TIMER_BEGIN(timer);
    {
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) 
            printf("before kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
        //else printf("before kernel launch success\n");
    }
    TIMER_PRINT("error check",timer);
    TIMER_BEGIN(timer);
    recordTime(ev2,ev3,"preprocessing 2");
    //minmaxKernel<<<blockPerGrid,threadsPerBlock>>>(d_resultMat,sss1.numSample,sss2.numSample,sss3.numSample,d_resultArray);
    minmaxKernelImproved<<<dimGrid,dimBlock>>>(d_resultMat,sss1.numSample,sss2.numSample,sss3.numSample,d_resultArray);
    recordTime(ev2,ev3,"min computing (kernel 2)");
    TIMER_PRINT("min kernel",timer);

    TIMER_BEGIN(timer);
    reportError();

    {
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) 
            printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
        //else printf("kernel launch success\n");
    }
    TIMER_PRINT("error check",timer);

    TIMER_BEGIN(timer);
    hipMemcpy(resultArray,d_resultArray,resultSize*sizeof(int),hipMemcpyDeviceToHost);
    TIMER_PRINT("post copy",timer);

    //cout<<sss1.numSample<<' '<<sss2.numSample<<endl;
    //for(int i=0;i<sss1.numSample;i++){
    //    cout<<i<<' '<<resultArray[i]<<endl;
    //}
    //cout<<endl;
    //cout<<resultArray[0]<<endl;

    TIMER_BEGIN(timer);
    
    for(int i=0;i<3;i++){
        hipFree(d_sssArray[i].labelArray);
        hipFree(d_sssArray[i].dataNodeOffsetArray);
        hipFree(d_sssArray[i].dataNodeIndexArray);
        hipFree(d_sssArray[i].dataNodeValueArray);
    }

    //hipFree(d_sss1);
    //hipFree(d_sss2);
    //hipFree(d_sss3);
    hipFree(d_resultMat);
    hipFree(d_test_data);
    hipFree(d_test_data_length);
    hipFree(d_resultArray);

    delete test_data;

    recordTime(ev2,ev3,"postprocessing");
    
    hipEventRecord(ev1,0);
    hipEventSynchronize(ev0);
    hipEventSynchronize(ev1);

    hipEventElapsedTime(&time,ev0,ev1);
    cout<<"cuda event time "<<time<<endl;

    hipEventDestroy(ev0);
    hipEventDestroy(ev1);
    hipEventDestroy(ev2);
    hipEventDestroy(ev3);

    TIMER_PRINT("post compute",timer);
    TIMER_PRINT("total compute time",timer0);
    delete [] resultMat;
    return resultArray;
}

float getDistance2(Data_Sample test_sample,const SerializedSampleSet sss,int i){
    int dataIndexBegin,dataIndexEnd;
    dataIndexBegin=i==0?0:sss.dataNodeOffsetArray[i-1];
    dataIndexEnd=sss.dataNodeOffsetArray[i];
    int j=0,k=dataIndexBegin;
    float sum=0;
    //cout<<endl<<dataIndexBegin<<endl<<dataIndexEnd<<endl;
    while(1){
        //cout<<j<<endl<<k<<endl;
        if(j==test_sample.data_vector_length && k==dataIndexEnd) break;
        if(k==dataIndexEnd) {
            sum+=SQUARE(test_sample.data_vector[j].value);
            j++;
        } else if(j==test_sample.data_vector_length) {
            sum+=SQUARE(sss.dataNodeValueArray[k]);
            k++;
        } else if(test_sample.data_vector[j].index < sss.dataNodeIndexArray[k]){
            sum+=SQUARE(test_sample.data_vector[j].value);
            j++;
        } else if(test_sample.data_vector[j].index > sss.dataNodeIndexArray[k]){
            sum+=SQUARE(sss.dataNodeValueArray[k]);
            k++;
        } else {
            float tmp=test_sample.data_vector[j].value-sss.dataNodeValueArray[k];
            sum+=SQUARE(tmp);
            //cout<<sum<<endl;
            j++;
            k++;
        }
    }
    return sum;
}

float getDistance2(const SerializedSampleSet sss1,int i,const SerializedSampleSet sss2,int j){
    int dataIndexBegin1,dataIndexEnd1;
    dataIndexBegin1=i==0?0:sss1.dataNodeOffsetArray[i-1];
    dataIndexEnd1=sss1.dataNodeOffsetArray[i];
    int dataIndexBegin2,dataIndexEnd2;
    dataIndexBegin2=j==0?0:sss2.dataNodeOffsetArray[j-1];
    dataIndexEnd2=sss2.dataNodeOffsetArray[j];
    int ii=dataIndexBegin1,ij=dataIndexBegin2;
    float sum=0;
    while (1){
        if(ii==dataIndexEnd1 && ij==dataIndexEnd2) break;
        else if(ii==dataIndexEnd1) {
            sum+=SQUARE(sss2.dataNodeValueArray[ij]);
            ij++;
        }else if (ij==dataIndexEnd2) {
            sum+=SQUARE(sss1.dataNodeValueArray[ii]);
            ii++;
        }else if (sss1.dataNodeIndexArray[ii]<sss2.dataNodeIndexArray[ij]){
            sum+=SQUARE(sss1.dataNodeValueArray[ii]);
            ii++;
        }else if (sss1.dataNodeIndexArray[ii]>sss2.dataNodeIndexArray[ij]){
            sum+=SQUARE(sss2.dataNodeValueArray[ij]);
            ij++;
        } else {
            float tmp=sss2.dataNodeValueArray[ij]-sss1.dataNodeValueArray[ii];
            sum+=SQUARE(tmp);
            ij++;
            ii++;
        } 
    }
    return sum;
}


int *m3gzcCPU(SerializedSampleSet sss1,SerializedSampleSet sss2,SerializedSampleSet sss3){
    //sss1.print();
    //sss2.print();

    float * sumArray0,* sumArray1,*sumArray2;
    int *resultArray;
    sumArray0=new float[sss2.numSample*sss1.numSample];
    sumArray1=new float[sss3.numSample*sss1.numSample];
    sumArray2=new float[sss3.numSample*sss2.numSample];
    resultArray=new int[sss3.numSample*sss1.numSample];
    
    Data_Node * test_data=new Data_Node;
    test_data->index=1;
    test_data->value=1;

    Data_Sample test_sample;
    test_sample.index=0;
    test_sample.label=0;
    test_sample.data_vector_length=1;
    test_sample.data_vector=test_data;

    //for(int i=0;i<sss1.numSample;i++) sumArray1[i]=getDistance2(test_sample,sss1,i);
    for(int i=0;i<sss1.numSample;i++) 
        for(int j=0;j<sss3.numSample;j++)
            sumArray1[j*sss1.numSample+i]=getDistance2(sss3,j,sss1,i);
    //cout<<sumArray1[0]<<endl;
    //for(int i=0;i<sss2.numSample;i++) sumArray2[i]=getDistance2(test_sample,sss2,i);
    for(int i=0;i<sss2.numSample;i++) 
        for(int j=0;j<sss3.numSample;j++)
            sumArray2[j*sss2.numSample+i]=getDistance2(sss3,j,sss2,i);

    
    for(int i=0;i<sss1.numSample;i++) 
        for(int j=0;j<sss2.numSample;j++)
            sumArray0[i*sss2.numSample+j]=getDistance2(sss2,j,sss1,i);

    for(int i=0;i<sss3.numSample;i++){
        for(int j=0;j<sss1.numSample;j++){
            float min=1;
            for(int k=0;k<sss2.numSample;k++){
                float sum0=sumArray0[j*sss2.numSample+k];
                float v=exp(-4*sumArray1[i*sss1.numSample+j]/sum0)-exp(-4*sumArray2[i*sss2.numSample+k]/sum0);
                if(v<min) min=v;
                //if(v<-THRESHOLD) break;
            }

            if(min>THRESHOLD) resultArray[i*sss1.numSample+j]=1;
            else if(min<-THRESHOLD) resultArray[i*sss1.numSample+j]=-1;
            else resultArray[i*sss1.numSample+j]=0;
        }
    }

    delete test_sample.data_vector;
    return resultArray;
}

int *m3gzcCPUOriginal(SerializedSampleSet sss1,SerializedSampleSet sss2,SerializedSampleSet sss3){
    clock_t timer;
    clock_t timer0;
    float time;
    TIMER_BEGIN(timer);
    TIMER_BEGIN(timer0);

    int * resultArray=new int[sss3.numSample*sss1.numSample];
        
    for(int i=0;i<sss3.numSample;i++){
        for(int j=0;j<sss1.numSample;j++){
            float min=1;
            for(int k=0;k<sss2.numSample;k++){
                float sum0=getDistance2(sss1,j,sss2,k);
                float sum1=getDistance2(sss1,j,sss3,i);
                float sum2=getDistance2(sss2,k,sss3,i);
                //float sum0=sumArray0[j*sss2.numSample+k];
                float v=exp(-4*sum1/sum0)-exp(-4*sum2/sum0);
                if(v<min) min=v;
                //if(v<-THRESHOLD) break;
            }

            if(min>THRESHOLD) resultArray[i*sss1.numSample+j]=1;
            else if(min<-THRESHOLD) resultArray[i*sss1.numSample+j]=-1;
            else resultArray[i*sss1.numSample+j]=0;
        }
    }
    TIMER_PRINT("Execution time",timer);
    return resultArray;
}
