#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<stdio.h>
#include"util.h"
#include"SerializedSampleSet.h"
#include"cuPrintf.cu"

using namespace std;

__device__ void print(int value){
    if(blockIdx.x==0 && blockIdx.y==0 && threadIdx.x==0 && threadIdx.y==1) 
        printf("block %d,%d, thread %d,%d, value %d\n",blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y,value);
}

__device__ void print(int x,int y,int value){
    if(blockIdx.x==0 && blockIdx.y==0 && threadIdx.x==x && threadIdx.y==y) 
        printf("block %d,%d, thread %d,%d, value %d\n",blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y,value);
}

__device__ void print(float value){
    if(blockIdx.x==0 && blockIdx.y==0 && threadIdx.x==0 && threadIdx.y==1) 
        printf("block %d,%d, thread %d,%d, value %f\n",blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y,value);
}

__device__ float getDistance(const Data_Node * data,const int dataLength,const int * dataNodeIndexArray,const float * dataNodeValueArray,const int dataNodeNum){
    float sum=0;
    float x=0,x1=0;
    int i=0,j=0;
    while(1){
        if (i==dataLength){ 
            for(;j<dataNodeNum;j++) sum+=dataNodeValueArray[j]*dataNodeValueArray[j];
            break;
        }
        else if(j==dataNodeNum) {
            for(;i<dataLength;i++) sum+=data[i].value*data[i].value;
            break;
        }
        x=data[i].value;
        x1=dataNodeValueArray[j];
        if(data[i].index<dataNodeIndexArray[j])  {x1=0; i++;}
        else if(data[i].index>dataNodeIndexArray[j]) {x=0; j++;}
        else {i++; j++;};
        float tmp=x-x1;
        sum+=tmp*tmp;
    }
    return sum;
}

__device__ float getDistance(const int * dataNodeIndexArray1,const float * dataNodeValueArray1,const int dataNodeNum1,const int * dataNodeIndexArray2,const float * dataNodeValueArray2,const int dataNodeNum2){
    float x1=0,x2=0;
    float sum=0;
    int loopcount=0;
    //printf("block %d,%d, thread %d,%d, value %d\n",blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y,dataNodeValueArray1[0]);
    //print(dataNodeNum1);
    //print(dataNodeIndexArray1[0]);
    //print(dataNodeValueArray1[0]);
    //print(dataNodeNum2);
    //print(dataNodeIndexArray2[0]);
    //print(dataNodeValueArray2[0]);
    int i=0,j=0;
    while(1){

        //print(i);
        //print(j);
        //print(loopcount);
        loopcount++; //if(loopcount==3) break;
        //if (i==dataNodeNum1){ 
        //    for(;j<dataNodeNum2;j++) sum+=dataNodeValueArray2[j]*dataNodeValueArray2[j];
        //    break;
        //}
        //else if(j==dataNodeNum2) {
        //    for(;i<dataNodeNum1;i++) sum+=dataNodeValueArray1[i]*dataNodeValueArray1[i];
        //    break;
        //}
        //x1=dataNodeValueArray1[i];
        //x2=dataNodeValueArray2[j];

        //int i1=-1,i2=-1;
        //if(i<dataNodeNum1) i1=dataNodeIndexArray1[i];
        //if(j<dataNodeNum2) i2=dataNodeIndexArray2[j];

        if (i==dataNodeNum1 && j==dataNodeNum2) {break;}
        else if((i<dataNodeNum1 && j==dataNodeNum2) || (j<dataNodeNum2 && i<dataNodeNum1 && dataNodeIndexArray1[i]<dataNodeIndexArray2[j]))  {
            x1=dataNodeValueArray1[j];
            x2=0; 
            i++;
        } else if((i==dataNodeNum1 && j<dataNodeNum2) || 
                (j<dataNodeNum2 && i<dataNodeNum1 && dataNodeIndexArray1[i]>dataNodeIndexArray2[j]) ){
            x2=dataNodeValueArray2[j];
            x1=0;
            j++;
        } else {
            x1=dataNodeValueArray1[i];
            x2=dataNodeValueArray2[j];
            i++; j++;
        }
        float tmp=x1-x2;
        sum+=tmp*tmp;
    }
    //print(sum);
    return sum;
}


__global__ void m3gzcKernel(const Data_Node * data,const int * dataLength,const SerializedSampleSet *sss1,const SerializedSampleSet *sss2,float * resultMat){
    int ix=blockIdx.x*blockDim.x+threadIdx.x;
    int iy=blockIdx.y*blockDim.y+threadIdx.y;
    float x=0,x1=0,x2=0,tmp=0;

    int lx=0,ly=0;
    if(ix>=sss1->numSample) return;
    if(iy>=sss2->numSample) return;
    int xbegin=ix>0?sss1->dataNodeOffsetArray[ix-1]:0;
    int ybegin=iy>0?sss2->dataNodeOffsetArray[iy-1]:0;
    int xend=sss1->dataNodeOffsetArray[ix];
    int yend=sss2->dataNodeOffsetArray[iy];

    float sum1=0,sum2=0,sum0=0;
    
    //print(ix);
    //print(iy);
    //cuPrintf("%d\n",3);
    //print(xbegin);
    //print(sss1->dataNodeValueArray[xbegin]);
    //print(ybegin);
    //print(sss2->dataNodeValueArray[ybegin]);
    //print(sss2->dataNodeValueArray[ybegin+1]);
    //print(sss2->dataNodeValueArray[ybegin+2]);
    sum0=getDistance(&(sss1->dataNodeIndexArray[xbegin]),&(sss1->dataNodeValueArray[xbegin]),xend-xbegin,&(sss2->dataNodeIndexArray[ybegin]),&(sss2->dataNodeValueArray[ybegin]),yend-ybegin);
    sum1=getDistance(data,*dataLength,&(sss1->dataNodeIndexArray[xbegin]),&(sss1->dataNodeValueArray[xbegin]),xend-xbegin);
    sum2=getDistance(data,*dataLength,&(sss2->dataNodeIndexArray[ybegin]),&(sss2->dataNodeValueArray[ybegin]),yend-ybegin);
    //print(sum0);
    //print(sum1);
    //print(sum2);

    float theta2=0.25*sum0;

    float result=expf(-sum1/theta2)-expf(-sum2/theta2);

    //print(result);

    resultMat[ix*(sss2->numSample)+iy]=result;

    return;
}
