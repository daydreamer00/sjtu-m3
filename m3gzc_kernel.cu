#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include"util.h"
#include"SerializedSampleSet.h"
#include"cuPrintf.cu"

using namespace std;

__device__ float getDistance(const Data_Node * data,const int dataLength,const int * dataNodeIndexArray,const float * dataNodeValueArray,const int dataNodeNum){
    float sum=0;
    float x=0,x1=0;
    while(1){
        int i=0,j=0;
        if (i==dataLength){ 
            for(;j<dataNodeNum;j++) sum+=dataNodeValueArray[j]*dataNodeValueArray[j];
            break;
        }
        else if(j==dataNodeNum) {
            for(;i<dataLength;i++) sum+=data[i].value*data[i].value;
            break;
        }
        x=data[i].value;
        x1=dataNodeValueArray[j];
        if(data[i].index<dataNodeIndexArray[j])  {x1=0; i++;}
        else if(data[i].index>dataNodeIndexArray[j]) {x=0; j++;}
        float tmp=x-x1;
        sum+=tmp*tmp;
    }
    return sum;
}

__device__ float getDistance(const int * dataNodeIndexArray1,const float * dataNodeValueArray1,const int dataNodeNum1,const int * dataNodeIndexArray2,const float * dataNodeValueArray2,const int dataNodeNum2){
    float x1=0,x2=0;
    float sum=0;
    cuPrintf("%d\n",dataNodeNum1);
    while(1){
        int i=0,j=0;
        if (i==dataNodeNum1){ 
            for(;j<dataNodeNum2;j++) sum+=dataNodeValueArray2[j]*dataNodeValueArray2[j];
            break;
        }
        else if(j==dataNodeNum2) {
            for(;i<dataNodeNum1;i++) sum+=dataNodeValueArray1[i]*dataNodeValueArray1[i];
            break;
        }
        x1=dataNodeValueArray1[j];
        x2=dataNodeValueArray2[j];
        if(dataNodeIndexArray1[i]<dataNodeIndexArray2[j])  {x2=0; i++;}
        else if(dataNodeIndexArray1[i]>dataNodeIndexArray2[j]) {x1=0; j++;}
        float tmp=x1-x2;
        sum+=tmp*tmp;
    }
    return sum;
}


__global__ void m3gzcKernel(const Data_Node * data,const int * dataLength,const SerializedSampleSet *sss1,const SerializedSampleSet *sss2,int * resultMat){
    int ix=blockIdx.x*blockDim.x+threadIdx.x;
    int iy=blockIdx.y*blockDim.y+threadIdx.y;
    float x=0,x1=0,x2=0,tmp=0;

    int lx=0,ly=0;
    if(ix>=sss1->numSample) return;
    if(iy>=sss2->numSample) return;
    int xbegin=ix>0?sss1->dataNodeOffsetArray[ix-1]:0;
    int ybegin=ix>0?sss2->dataNodeOffsetArray[iy-1]:0;
    int xend=sss1->dataNodeOffsetArray[ix];
    int yend=sss2->dataNodeOffsetArray[iy];

    float sum1=0,sum2=0,sum0=0;
    
    sum0=getDistance(&(sss1->dataNodeIndexArray[xbegin]),&(sss1->dataNodeValueArray[xbegin]),xend-xbegin,&(sss2->dataNodeIndexArray[ybegin]),&(sss2->dataNodeValueArray[ybegin]),yend-ybegin);
    //sum1=getDistance(data,*dataLength,&(sss1->dataNodeIndexArray[xbegin]),&(sss1->dataNodeValueArray[xbegin]),xend-xbegin);
    //sum2=getDistance(data,*dataLength,&(sss2->dataNodeIndexArray[ybegin]),&(sss2->dataNodeValueArray[ybegin]),yend-ybegin);

    float theta2=0.25*sum0;
    

    float result=exp10f(-sum1/theta2)-exp10f(-sum2/theta2);

    resultMat[ix*(SerializedSampleSet::max_num_sample)+iy]=result;

    return;
}
