#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<stdio.h>
#include"util.h"
#include"SerializedSampleSet.h"
#include"cuPrintf.cu"

#define BLOCK_SIZE 16
#define AVERAGE_DATA_PER_SAMPLE 100
#define SQUARE(x) (x*x)

using namespace std;

__device__ void print(int value){
    if(blockIdx.x==0 && blockIdx.y==0 && threadIdx.x==0 && threadIdx.y==1) 
        printf("block %d,%d, thread %d,%d, value %d\n",blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y,value);
}

__device__ void print(int x,int y,int value){
    if(blockIdx.x==0 && blockIdx.y==0 && threadIdx.x==x && threadIdx.y==y) 
        printf("block %d,%d, thread %d,%d, value %d\n",blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y,value);
}

__device__ void print(int x,int y,char * value){
    //if(blockIdx.x==0 && blockIdx.y==0 && threadIdx.x==x && threadIdx.y==y) 
        printf("block %d,%d, thread %d,%d, value %s\n",blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y,value);
}

__device__ void print(float value){
    if(blockIdx.x==0 && blockIdx.y==0 && threadIdx.x==0 && threadIdx.y==1) 
        printf("block %d,%d, thread %d,%d, value %f\n",blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y,value);
}


__device__ float getDistance(const Data_Node * data,const int dataLength,const int * dataNodeIndexArray,const float * dataNodeValueArray,const int dataNodeNum){
    float sum=0;
    float x=0,x1=0;
    int i=0,j=0;
    while(1){
        if (i==dataLength){ 
            for(;j<dataNodeNum;j++) sum+=dataNodeValueArray[j]*dataNodeValueArray[j];
            break;
        }
        else if(j==dataNodeNum) {
            for(;i<dataLength;i++) sum+=data[i].value*data[i].value;
            break;
        }
        x=data[i].value;
        x1=dataNodeValueArray[j];
        if(data[i].index<dataNodeIndexArray[j])  {x1=0; i++;}
        else if(data[i].index>dataNodeIndexArray[j]) {x=0; j++;}
        else {i++; j++;};
        float tmp=x-x1;
        sum+=tmp*tmp;
    }
    return sum;
}

__device__ float getDistance(const int * dataNodeIndexArray1,const float * dataNodeValueArray1,const int dataNodeNum1,const int * dataNodeIndexArray2,const float * dataNodeValueArray2,const int dataNodeNum2){
    float x1=0,x2=0;
    float sum=0;
    int loopcount=0;
    //printf("block %d,%d, thread %d,%d, value %d\n",blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y,dataNodeValueArray1[0]);
    //print(dataNodeNum1);
    //print(dataNodeIndexArray1[0]);
    //print(dataNodeValueArray1[0]);
    //print(dataNodeNum2);
    //print(dataNodeIndexArray2[0]);
    //print(dataNodeValueArray2[0]);
    int i=0,j=0;
    while(1){

        //print(i);
        //print(j);
        //print(loopcount);
        loopcount++; //if(loopcount==3) break;
        //if (i==dataNodeNum1){ 
        //    for(;j<dataNodeNum2;j++) sum+=dataNodeValueArray2[j]*dataNodeValueArray2[j];
        //    break;
        //}
        //else if(j==dataNodeNum2) {
        //    for(;i<dataNodeNum1;i++) sum+=dataNodeValueArray1[i]*dataNodeValueArray1[i];
        //    break;
        //}
        //x1=dataNodeValueArray1[i];
        //x2=dataNodeValueArray2[j];

        //int i1=-1,i2=-1;
        //if(i<dataNodeNum1) i1=dataNodeIndexArray1[i];
        //if(j<dataNodeNum2) i2=dataNodeIndexArray2[j];

        if (i==dataNodeNum1 && j==dataNodeNum2) {break;}
        else if((i<dataNodeNum1 && j==dataNodeNum2) || (j<dataNodeNum2 && i<dataNodeNum1 && dataNodeIndexArray1[i]<dataNodeIndexArray2[j]))  {
            x1=dataNodeValueArray1[j];
            x2=0; 
            i++;
        } else if((i==dataNodeNum1 && j<dataNodeNum2) || 
                (j<dataNodeNum2 && i<dataNodeNum1 && dataNodeIndexArray1[i]>dataNodeIndexArray2[j]) ){
            x2=dataNodeValueArray2[j];
            x1=0;
            j++;
        } else {
            x1=dataNodeValueArray1[i];
            x2=dataNodeValueArray2[j];
            i++; j++;
        }
        float tmp=x1-x2;
        sum+=tmp*tmp;
    }
    //print(sum);
    return sum;
}


__device__ void loadToSharedMemory(int ix0,int ix1,int iy0,int iy1,
        int xbegin,int xend,int ybegin,int yend,
        int ix,int iy,const SerializedSampleSet *sss1,const SerializedSampleSet *sss2,
        int *dataOffsetArray1,int *dataIndexArray1,float *dataValueArray1,
        int *dataOffsetArray2,int *dataIndexArray2,float *dataValueArray2){

    if(threadIdx.y==blockDim.y-1) dataOffsetArray1[threadIdx.x]=sss1->dataNodeOffsetArray[ix];
    if(threadIdx.x==blockDim.x-1) dataOffsetArray2[threadIdx.y]=sss2->dataNodeOffsetArray[iy];

    //These variables should be in shared memory
    //int ix0,iy0,ix1,iy1;
    //int xbegin,ybegin,xend,yend;

    //if(threadIdx.x==0 && threadIdx.y==0){
    //This block of code needs only execute once on a block
        //int ix0=blockIdx.x*blockDim.x,iy0=blockIdx.y*blockIdx.y;
        //int ix1=blockIdx.x<gridDim.x-1?ix0+blockDim.x-1:sss1->numSample-1;
        //int iy1=blockIdx.y<gridDim.y-1?iy0+blockDim.y-1:sss2->numSample-1;
        //int xbegin=ix0>0?sss1->dataNodeOffsetArray[ix0-1]:0;
        //int ybegin=iy0>0?sss2->dataNodeOffsetArray[iy0-1]:0;
        //int xend=sss1->dataNodeOffsetArray[ix1];
        //int yend=sss2->dataNodeOffsetArray[iy1];
    //}

    //print(xend);

    if(xend-xbegin>BLOCK_SIZE*AVERAGE_DATA_PER_SAMPLE || yend-ybegin>BLOCK_SIZE*AVERAGE_DATA_PER_SAMPLE)
        print(0,0,"too many data");

    if(xend-xbegin>0) 
        for(int i=0;i<(xend-xbegin-1)/(blockDim.x*blockDim.y)+1;i++) {
            int j=i*blockDim.x*blockDim.y+threadIdx.x*blockDim.y+threadIdx.y;
            if(j+xbegin<xend) {
                dataIndexArray1[j]=sss1->dataNodeIndexArray[xbegin+j];
                dataValueArray1[j]=sss1->dataNodeValueArray[xbegin+j];
            }
        }
    if(yend-ybegin>0) 
        for(int i=0;i<(yend-ybegin-1)/(blockDim.x*blockDim.y)+1;i++) {
            int j=ybegin+i*blockDim.x*blockDim.y+threadIdx.x*blockDim.y+threadIdx.y;
            if(j+ybegin<yend) {
                dataIndexArray2[j]=sss2->dataNodeIndexArray[ybegin+j];
                dataValueArray2[j]=sss2->dataNodeValueArray[ybegin+j];
            }
        }
}


__global__ void m3gzcKernel(const Data_Node * data,const int * dataLength,const SerializedSampleSet *sss1,const SerializedSampleSet *sss2,float * resultMat){
    int ix=blockIdx.x*blockDim.x+threadIdx.x;
    int iy=blockIdx.y*blockDim.y+threadIdx.y;
    float x=0,x1=0,x2=0,tmp=0;

    if(ix>=sss1->numSample) return;
    if(iy>=sss2->numSample) return;
    int xbegin=ix>0?sss1->dataNodeOffsetArray[ix-1]:0;
    int ybegin=iy>0?sss2->dataNodeOffsetArray[iy-1]:0;
    int xend=sss1->dataNodeOffsetArray[ix];
    int yend=sss2->dataNodeOffsetArray[iy];

    float sum1=0,sum2=0,sum0=0;
    
    //print(ix);
    //print(iy);
    //cuPrintf("%d\n",3);
    //print(xbegin);
    //print(sss1->dataNodeValueArray[xbegin]);
    //print(ybegin);
    //print(sss2->dataNodeValueArray[ybegin]);
    //print(sss2->dataNodeValueArray[ybegin+1]);
    //print(sss2->dataNodeValueArray[ybegin+2]);
    sum0=getDistance(&(sss1->dataNodeIndexArray[xbegin]),&(sss1->dataNodeValueArray[xbegin]),xend-xbegin,&(sss2->dataNodeIndexArray[ybegin]),&(sss2->dataNodeValueArray[ybegin]),yend-ybegin);
    sum1=getDistance(data,*dataLength,&(sss1->dataNodeIndexArray[xbegin]),&(sss1->dataNodeValueArray[xbegin]),xend-xbegin);
    sum2=getDistance(data,*dataLength,&(sss2->dataNodeIndexArray[ybegin]),&(sss2->dataNodeValueArray[ybegin]),yend-ybegin);
    //print(sum0);
    //print(sum1);
    //print(sum2);

    float theta2=0.25*sum0;

    float result=expf(-sum1/theta2)-expf(-sum2/theta2);

    //print(result);

    resultMat[ix*(sss2->numSample)+iy]=result;

    return;
}

__global__ void m3gzcKernelWithSharedMemory(const Data_Node * data,const int * dataLength,const SerializedSampleSet *sss1,const SerializedSampleSet *sss2,float * resultMat){

    __shared__ float sum1[BLOCK_SIZE];
    __shared__ float sum2[BLOCK_SIZE];
    __shared__ int dataOffsetArray1[BLOCK_SIZE]; 
    __shared__ int dataOffsetArray2[BLOCK_SIZE]; 
    __shared__ int dataIndexArray1[BLOCK_SIZE*AVERAGE_DATA_PER_SAMPLE]; 
    __shared__ int dataIndexArray2[BLOCK_SIZE*AVERAGE_DATA_PER_SAMPLE]; 
    __shared__ float dataValueArray1[BLOCK_SIZE*AVERAGE_DATA_PER_SAMPLE]; 
    __shared__ float dataValueArray2[BLOCK_SIZE*AVERAGE_DATA_PER_SAMPLE]; 

    __shared__ int ixBlockBegin,ixBlockEnd,iyBlockBegin,iyBlockEnd;
    __shared__ int xBlockBegin,xBlockEnd,yBlockBegin,yBlockEnd;

    //__shared__ float dataValueArray3[1200000]; 
    //dataValueArray3[1199999]=0;
    
    if(threadIdx.x==0 && threadIdx.y==0){
        ixBlockBegin=blockIdx.x*blockDim.x;
        iyBlockBegin=blockIdx.y*blockIdx.y;
        ixBlockEnd=blockIdx.x<gridDim.x-1?ixBlockBegin+blockDim.x-1:sss1->numSample-1;
        iyBlockEnd=blockIdx.y<gridDim.y-1?iyBlockBegin+blockDim.y-1:sss2->numSample-1;
        xBlockBegin=ixBlockBegin>0?sss1->dataNodeOffsetArray[ixBlockBegin-1]:0;
        yBlockBegin=iyBlockBegin>0?sss2->dataNodeOffsetArray[iyBlockBegin-1]:0;
        xBlockEnd=sss1->dataNodeOffsetArray[ixBlockEnd];
        yBlockEnd=sss2->dataNodeOffsetArray[iyBlockEnd];
    }

    int ix=blockIdx.x*blockDim.x+threadIdx.x;
    int iy=blockIdx.y*blockDim.y+threadIdx.y;
    float x=0,x1=0,x2=0,tmp=0;

    loadToSharedMemory(ixBlockBegin,ixBlockEnd,iyBlockBegin,iyBlockEnd,
            xBlockBegin,xBlockEnd,yBlockBegin,yBlockEnd,
            ix,iy,sss1,sss2,
            dataOffsetArray1,dataIndexArray1,dataValueArray1,
            dataOffsetArray2,dataIndexArray2,dataValueArray2);

    __syncthreads();
    //for(int i=0;i<BLOCK_SIZE;i++) print(dataIndexArray2[i]);

    if(ix>=sss1->numSample) return;
    if(iy>=sss2->numSample) return;

    int xbegin=ix>0?dataOffsetArray1[ix-1]:0;
    int ybegin=iy>0?dataOffsetArray2[iy-1]:0;
    int xend=dataOffsetArray1[ix];
    int yend=dataOffsetArray2[iy];

    if(threadIdx.y==0)
        sum1[threadIdx.x]=getDistance(data,*dataLength,&(dataIndexArray1[xbegin]),&(dataValueArray1[xbegin]),xend-xbegin);
    if(threadIdx.x==0)
        sum2[threadIdx.y]=getDistance(data,*dataLength,&(dataIndexArray2[ybegin]),&(dataValueArray2[ybegin]),yend-ybegin);

    __syncthreads();

    float sum0=0;
    //float sum1=0,sum2=0;
    
    //print(ix);
    //print(iy);
    //cuPrintf("%d\n",3);
    //print(xbegin);
    //print(sss1->dataNodeValueArray[xbegin]);
    //print(ybegin);
    //print(sss2->dataNodeValueArray[ybegin]);
    //print(sss2->dataNodeValueArray[ybegin+1]);
    //print(sss2->dataNodeValueArray[ybegin+2]);

    sum0=getDistance(&(dataIndexArray1[xbegin]),&(dataValueArray1[xbegin]),xend-xbegin,&(dataIndexArray2[ybegin]),&(dataValueArray2[ybegin]),yend-ybegin);
    
    //print(sum0);
    //print(sum1);
    //print(sum2);

    float theta2=0.25*sum0;

    float result;
    result=expf(-sum1[threadIdx.x]/theta2)-expf(-sum2[threadIdx.y]/theta2);

    //print(result);

    resultMat[ix*(sss2->numSample)+iy]=result;

    return;
}
